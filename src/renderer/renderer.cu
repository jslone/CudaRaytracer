#include "hip/hip_runtime.h"
#include "renderer.h"
#include <cstring>
#include <iostream>
#include <cuda_gl_interop.h>

namespace acr
{
	struct DevParams
	{
		char sceneData[sizeof(Scene)];
		uint32_t width,height,samples;
	};

	__constant__
	DevParams devParams[1];
	
	Renderer *renderer;
	void globalRender()
	{
		renderer->render();
	}

	Renderer::Renderer(const Renderer::Args &args)
		: title(args.title)
		, dim(args.dim)
	{
		renderer = this;

		/* Create window */
		glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
		glutInitWindowSize(dim.x, dim.y);

		winId = glutCreateWindow(title);

		GLenum err = glewInit();
		if (err != GLEW_OK)
		{
			std::cerr << "glewInit: " << glewGetErrorString(err) << std::endl;
			exit(EXIT_FAILURE);
		}

		glutDisplayFunc(globalRender);

		/* Set the clear color. */
		glClearColor( 0, 0, 0, 1 );
		glClear(GL_COLOR_BUFFER_BIT);

		/* Setup our viewport. */
		glViewport( 0, 0, dim.x, dim.y );
		
		/* Setup the projection and world matrix */
		glMatrixMode( GL_PROJECTION );
		glLoadIdentity( );

		glOrtho(0,1.0f,0,1.0f,-1.0f,1.0f);
		glMatrixMode(GL_MODELVIEW);
		glLoadIdentity();

		// cuda interop initialization
		uint32_t numDevices;
		const uint32_t maxNumDevices = 1;
		int devices[maxNumDevices];
		hipError_t cudaErr = hipGLGetDevices(&numDevices, devices, maxNumDevices, hipGLDeviceListAll);
		if(cudaErr != hipSuccess)
		{
			std::cout << "hipGLGetDevices[" << cudaErr << "]: ";
			switch(cudaErr)
			{
				case hipErrorNoDevice:
					std::cout << "No device found." << std::endl;
					break;
				default:
					std::cout << "Error unknown." << std::endl;
			}
			exit(EXIT_FAILURE);
		}

		cudaErr = cudaGLSetGLDevice(devices[0]);
		if(cudaErr != hipSuccess)
		{
			std::cout << "cudaGLSetGLDevice: ";
			switch(cudaErr)
			{
				case hipErrorInvalidDevice:
					std::cout << "Invalid device." << std::endl;
					break;
				default:
					std::cout << "Device already set." << std::endl;
			}
			exit(EXIT_FAILURE);
		}

		// setup draw buffer
		glGenBuffers(1,&drawBuffer);

		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, drawBuffer);

		glBufferData(GL_PIXEL_UNPACK_BUFFER, dim.x * dim.y * sizeof(Color4), NULL, GL_DYNAMIC_COPY);

		cudaGLRegisterBufferObject(drawBuffer);

		// setup texture
		glEnable(GL_TEXTURE_2D);
		glGenTextures(1,&textureId);
		
		glBindTexture(GL_TEXTURE_2D, textureId);

		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, dim.x, dim.y, 0, GL_RGBA, GL_FLOAT, nullptr);

		glTexParameteri(GL_TEXTURE_2D,GL_TEXTURE_MIN_FILTER,GL_LINEAR);
		glTexParameteri(GL_TEXTURE_2D,GL_TEXTURE_MAG_FILTER,GL_LINEAR);

		hipMalloc((void**)&cuRandStates, sizeof(hiprandState) * dim.x * dim.y);
	}

	Renderer::~Renderer()
	{
		glutDestroyWindow(winId);
	}

	void Renderer::moveCamera(const math::vec3 &pos, const math::vec2 &dir)
	{
		std::cout << "Rot amount: " << math::to_string(dir) << std::endl;
		DevParams param;
		hipMemcpyFromSymbol(&param, HIP_SYMBOL(devParams), sizeof(DevParams));

		Scene *scene = (Scene*)&param;

		scene->camera.forward = math::rotate(scene->camera.forward, dir.x, scene->camera.up);

		math::vec3 right = math::cross(scene->camera.forward, scene->camera.up);

		scene->camera.forward = math::rotate(scene->camera.forward, dir.y, right);
		scene->camera.up = math::rotate(scene->camera.up, dir.y, right);

		scene->camera.position += pos;

		std::cout << "New Dir: " << math::to_string(scene->camera.forward) << std::endl;

		hipMemcpyToSymbol(HIP_SYMBOL(devParams), &param, sizeof(DevParams));
	}

	void Renderer::loadScene(const Scene &scene)
	{
		DevParams params;
		std::memcpy(params.sceneData, &scene, sizeof(Scene));
		params.width = dim.x;
		params.height = dim.y;
		params.samples = dim.z;

		hipMemcpyToSymbol(HIP_SYMBOL(devParams), &params, sizeof(DevParams));
	}

	__device__
	math::vec3 get_pixel_dir(const Camera &camera, float ni, float nj)
	{

		math::vec3 dir;
		math::vec3 up;
		float AR;

		math::vec3 cR;
		math::vec3 cU;
		float dist;
		math::vec3 pos;
    
		dir = camera.forward;
		up = camera.up;
		AR = camera.aspectRatio;
		cR = math::cross(dir, up);
		cU = math::cross(cR, dir);
		pos = camera.position;
		dist = math::fastertanfull(camera.horizontalFOV/2.0f);
		
		return math::normalize(dir + dist*(float(nj)*cU + AR*float(ni)*cR));
	}

	__global__
	void scatterTrace(Color4 *screen/*, hiprandState *randState, unsigned long seed*/)
	{
		const int width = devParams->width;
		const int height = devParams->height;

		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
		int index = x + width * y;

		if (x >= width || y >= height)
		{
			return;
		}
		
		
		//hiprand_init(seed,index,0,&randState[index]);

		Scene *scene = (Scene*)devParams;

		float dx = 1.0f / width;
		float dy = 1.0f / height;
		
		float i = 2.0f*(float(x) + 0.5f)*dx - 1.0f;//2.0f*(float(x)+hiprand_uniform(&randState[index]))*dx - 1.0f;
		float j = 2.0f*(float(y) + 0.5f)*dy - 1.0f;//2.0f*(float(y) + hiprand_uniform(&randState[index]))*dy - 1.0f;

		
		Ray r;
		r.o = scene->camera.position;
		r.d = get_pixel_dir(scene->camera, i, j);
		
		if (false && x == width - 1 && y == height - 1)
		{
			printf("pos: (%f, %f, %f), dir: (%f, %f, %f)\n", r.o.x, r.o.y, r.o.z, r.d.x, r.d.y, r.d.z);
			for (int i = 0; i < scene->materials.size(); i++)
			{
				Color3 &c = scene->materials[i].diffuse;
				printf("Diffuse: %f, %f, %f, %f\n", c.r, c.g, c.b);
			}
		}

		HitInfo info;
		info.t = FLT_MAX;
		Color4 contribution = Color4(1, 0, 0, 1);
		
		if(scene->intersect(r,info))
		{
			Color3 &c = scene->materials[info.materialIndex].diffuse;
			contribution = Color4(c, 1);
		}

		screen[index] = contribution;
	}

	void Renderer::render()
	{
		// bind draw buffer to device ptr
		Color4 *screen;
		hipError_t err = cudaGLMapBufferObject((void**)&screen, drawBuffer);
		if (err != hipSuccess)
		{
			std::cerr << "cudaGLMapBufferObject: " << hipGetErrorName(err) << std::endl;
		}

		// call kernel to render pixels then draw to screen
		dim3 block(16,16);
		dim3 grid((dim.x + block.x - 1) / block.x, (dim.y + block.y - 1) / block.y);

		scatterTrace<<<grid,block>>>(screen/*,cuRandStates,glutGet(GLUT_ELAPSED_TIME)*/);
		hipDeviceSynchronize();

		// unbind draw buffer so openGL can use
		cudaGLUnmapBufferObject(drawBuffer);

		// create texture from draw buffer
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, drawBuffer);
		glActiveTexture(GL_TEXTURE0);
		glBindTexture(GL_TEXTURE_2D, textureId);

		glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, dim.x, dim.y, GL_RGBA, GL_FLOAT, nullptr);
		GLenum glErr = glGetError();
		if (glErr != GL_NO_ERROR)
		{
			std::cerr << "glTexImage2D: " << gluErrorString(glErr) << std::endl;
		}

		// draw fullscreen quad
		glBegin(GL_QUADS);
			glTexCoord2f( 0, 1.0f);
			glVertex3f(0,0,0);
			glTexCoord2f(0,0);
			glVertex3f(0,1.0f,0);
			glTexCoord2f(1.0f,0);
			glVertex3f(1.0f,1.0f,0);
			glTexCoord2f(1.0f,1.0f);
			glVertex3f(1.0f,0,0);
		glEnd();
		
		// swap buffers
		glutSwapBuffers();
		glutPostRedisplay();
	}

}
