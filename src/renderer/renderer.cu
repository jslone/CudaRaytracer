#include "hip/hip_runtime.h"
#include "renderer.h"
#include <cstring>
#include <iostream>
#include <cuda_gl_interop.h>

namespace acr
{
	struct DevParams
	{
		char sceneData[sizeof(Scene)];
		uint32_t width,height,samples;
	};

	__constant__
	DevParams devParams[1];
	
	Renderer *renderer;
	void globalRender()
	{
		renderer->render();
	}

	Renderer::Renderer(const Renderer::Args &args)
		: title(args.title)
		, dim(args.dim)
	{
		renderer = this;

		/* Create window */
		glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
		glutInitWindowSize(dim.x, dim.y);

		dim.x *= dim.z;
		dim.y *= dim.z;
		dim.z = 1;

		winId = glutCreateWindow(title);

		GLenum err = glewInit();
		if (err != GLEW_OK)
		{
			std::cerr << "glewInit: " << glewGetErrorString(err) << std::endl;
			exit(EXIT_FAILURE);
		}

		glutDisplayFunc(globalRender);

		/* Set the clear color. */
		glClearColor( 0, 0, 0, 1 );
		glClear(GL_COLOR_BUFFER_BIT);

		/* Setup our viewport. */
		glViewport( 0, 0, dim.x, dim.y );
		
		/* Setup the projection and world matrix */
		glMatrixMode( GL_PROJECTION );
		glLoadIdentity( );

		glOrtho(0,1.0f,0,1.0f,-1.0f,1.0f);
		glMatrixMode(GL_MODELVIEW);
		glLoadIdentity();

		// cuda interop initialization
		uint32_t numDevices;
		const uint32_t maxNumDevices = 1;
		int devices[maxNumDevices];
		hipError_t cudaErr = hipGLGetDevices(&numDevices, devices, maxNumDevices, hipGLDeviceListAll);
		if(cudaErr != hipSuccess)
		{
			std::cout << "hipGLGetDevices[" << cudaErr << "]: ";
			switch(cudaErr)
			{
				case hipErrorNoDevice:
					std::cout << "No device found." << std::endl;
					break;
				default:
					std::cout << "Error unknown." << std::endl;
			}
			exit(EXIT_FAILURE);
		}

		cudaErr = cudaGLSetGLDevice(devices[0]);
		if(cudaErr != hipSuccess)
		{
			std::cout << "cudaGLSetGLDevice: ";
			switch(cudaErr)
			{
				case hipErrorInvalidDevice:
					std::cout << "Invalid device." << std::endl;
					break;
				default:
					std::cout << "Device already set." << std::endl;
			}
			exit(EXIT_FAILURE);
		}

		// setup draw buffer
		glGenBuffers(1,&drawBuffer);

		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, drawBuffer);

		glBufferData(GL_PIXEL_UNPACK_BUFFER, dim.x * dim.y * sizeof(Color4), NULL, GL_DYNAMIC_COPY);

		cudaGLRegisterBufferObject(drawBuffer);

		// setup texture
		glEnable(GL_TEXTURE_2D);
		glGenTextures(1,&textureId);
		
		glBindTexture(GL_TEXTURE_2D, textureId);

		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, dim.x, dim.y, 0, GL_RGBA, GL_FLOAT, nullptr);

		glTexParameteri(GL_TEXTURE_2D,GL_TEXTURE_MIN_FILTER,GL_LINEAR);
		glTexParameteri(GL_TEXTURE_2D,GL_TEXTURE_MAG_FILTER,GL_LINEAR);
	}

	Renderer::~Renderer()
	{
		glutDestroyWindow(winId);
	}

	void Renderer::moveCamera(const math::vec2 &pos, const math::vec2 &dir)
	{
		DevParams param;
		hipMemcpyFromSymbol(&param, HIP_SYMBOL(devParams), sizeof(DevParams));

		Scene *scene = (Scene*)&param;

		scene->camera.forward = math::rotate(scene->camera.forward, dir.x, scene->camera.up);

		math::vec3 right = math::cross(scene->camera.forward, scene->camera.up);

		scene->camera.forward = math::rotate(scene->camera.forward, dir.y, right);
		scene->camera.up = math::rotate(scene->camera.up, dir.y, right);

		math::vec3 delta = scene->camera.forward * pos.y + right * pos.x;

		scene->camera.position += delta;

		hipMemcpyToSymbol(HIP_SYMBOL(devParams), &param, sizeof(DevParams));
	}

	void Renderer::loadScene(const Scene &scene)
	{
		DevParams params;
		std::memcpy(params.sceneData, &scene, sizeof(Scene));
		params.width = dim.x;
		params.height = dim.y;
		params.samples = dim.z;

		Scene *myScene = (Scene*)&params;
		myScene->camera.aspectRatio = float(dim.x) / float(dim.y);

		hipMemcpyToSymbol(HIP_SYMBOL(devParams), &params, sizeof(DevParams));
	}

	__device__
	math::vec3 get_pixel_dir(const Camera &camera, float ni, float nj)
	{

		math::vec3 dir;
		math::vec3 up;
		float AR;

		math::vec3 cR;
		math::vec3 cU;
		float dist;
		math::vec3 pos;
    
		dir = camera.forward;
		up = camera.up;
		AR = camera.aspectRatio;
		cR = math::cross(dir, up);
		cU = math::cross(cR, dir);
		pos = camera.position;
		dist = math::fastertanfull(camera.horizontalFOV/2.0f);
		
		return math::normalize(dir + dist*(float(nj)*cU + AR*float(ni)*cR));
	}

	__device__
	math::vec3 get_pixel_pos(const Camera &camera, float ni, float nj)
	{

		math::vec3 dir;
		math::vec3 up;
		float AR;

		math::vec3 cR;
		math::vec3 cU;
		float dist;
		math::vec3 pos;

		dir = camera.forward;
		up = camera.up;
		AR = camera.aspectRatio;
		cR = math::cross(dir, up);
		cU = math::cross(cR, dir);
		pos = camera.position;
		dist = math::fastertanfull(camera.horizontalFOV / 2.0f);

		return camera.position + dist*(float(nj)*cU + AR*float(ni)*cR);
	}

	__global__
	void scatterTrace(Color4 *screen, unsigned long seed)
	{
		const int width = devParams->width;
		const int height = devParams->height;

		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
		int index = x + width * y;

		if (x >= width || y >= height)
		{
			return;
		}
		
		hiprandState state;
		hiprand_init(seed, 0, 0, &state);

		Scene *scene = (Scene*)devParams;

		float dx = 1.0f / width;
		float dy = 1.0f / height;
		
		float i = 2.0f*(float(x) + hiprand_uniform(&state))*dx - 1.0f;
		float j = 2.0f*(float(y) + hiprand_uniform(&state))*dy - 1.0f;

		Ray r;
		r.o = scene->camera.position;
		r.d = get_pixel_dir(scene->camera, i, -j);

		HitInfo info;
		info.t = FLT_MAX;
		Color4 contribution = Color4(0, 0, 0, 1);
		if(scene->intersect(r,info))
		{
			Material &mat = scene->materials[info.materialIndex];
			Color3 c = mat.ambient
				+ mat.diffuse * scene->lightPoint(info.point.position, info.point.normal);
			contribution = Color4(c, 1);
			
			if (x == width / 2 && y == height / 2)
			{
				//printf("Pos: (%f,%f,%f), Norm: (%f,%f,%f)\n", info.point.position.x, info.point.position.y, info.point.position.z, info.point.normal.x, info.point.normal.y, info.point.normal.z);
			}
			//contribution = Color4(info.point.position / Color3(-6,6,6), 1); // render position
			//contribution = Color4((info.point.normal + Color3(1,1,1)) / 2.0f, 1); // render normals
		}

		screen[index] = contribution;
	}

	void Renderer::render()
	{
		// bind draw buffer to device ptr
		Color4 *screen;
		hipError_t err = cudaGLMapBufferObject((void**)&screen, drawBuffer);
		if (err != hipSuccess)
		{
			std::cerr << "cudaGLMapBufferObject: " << hipGetErrorName(err) << std::endl;
		}

		// call kernel to render pixels then draw to screen
		dim3 block(16,16);
		dim3 grid((dim.x + block.x - 1) / block.x, (dim.y + block.y - 1) / block.y);

		scatterTrace<<<grid,block>>>(screen,glutGet(GLUT_ELAPSED_TIME));
		hipDeviceSynchronize();

		// unbind draw buffer so openGL can use
		cudaGLUnmapBufferObject(drawBuffer);

		// create texture from draw buffer
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, drawBuffer);
		glActiveTexture(GL_TEXTURE0);
		glBindTexture(GL_TEXTURE_2D, textureId);

		glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, dim.x, dim.y, GL_RGBA, GL_FLOAT, nullptr);
		GLenum glErr = glGetError();
		if (glErr != GL_NO_ERROR)
		{
			std::cerr << "glTexImage2D: " << gluErrorString(glErr) << std::endl;
		}

		// draw fullscreen quad
		glBegin(GL_QUADS);
			glTexCoord2f( 0, 1.0f);
			glVertex3f(0,0,0);
			glTexCoord2f(0,0);
			glVertex3f(0,1.0f,0);
			glTexCoord2f(1.0f,0);
			glVertex3f(1.0f,1.0f,0);
			glTexCoord2f(1.0f,1.0f);
			glVertex3f(1.0f,0,0);
		glEnd();
		
		// swap buffers
		glutSwapBuffers();
		glutPostRedisplay();
	}

}
