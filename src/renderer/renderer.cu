#include "hip/hip_runtime.h"
#include "renderer.h"
#include <cstring>
#include <iostream>
#include <cuda_gl_interop.h>
#include <string>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>

#define MAX_BOUNCES 5

namespace acr
{ 
	struct DevParams
	{
		char sceneData[sizeof(Scene)];
		uint32_t width,height,samples;\
		PathAggregate* paths;
		int* pixelValues;
	};

	__constant__
	DevParams devParams[1];

	Renderer *renderer;
	void globalRender()
	{
		renderer->render();
	}

	Renderer::Renderer(const Renderer::Args &args)
		: title(args.title)
		, dim(args.dim)
		, framesNoMove(0)
	{
		renderer = this;

		/* Create window */
		glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
		glutInitWindowSize(dim.x, dim.y);

		dim.x *= dim.z;
		dim.y *= dim.z;
		dim.z = 1;

		winId = glutCreateWindow(title);

		GLenum err = glewInit();
		if (err != GLEW_OK)
		{
			std::cerr << "glewInit: " << glewGetErrorString(err) << std::endl;
			exit(EXIT_FAILURE);
		}

		glutDisplayFunc(globalRender);

		/* Set the clear color. */
		glClearColor( 0, 0, 0, 1 );
		glClear(GL_COLOR_BUFFER_BIT);

		/* Setup our viewport. */
		glViewport( 0, 0, dim.x, dim.y );
		
		/* Setup the projection and world matrix */
		glMatrixMode( GL_PROJECTION );
		glLoadIdentity( );

		glOrtho(0,1.0f,0,1.0f,-1.0f,1.0f);
		glMatrixMode(GL_MODELVIEW);
		glLoadIdentity();

		// cuda interop initialization
		uint32_t numDevices;
		const uint32_t maxNumDevices = 1;
		int devices[maxNumDevices];
		hipError_t cudaErr = hipGLGetDevices(&numDevices, devices, maxNumDevices, hipGLDeviceListAll);
		if(cudaErr != hipSuccess)
		{
			std::cout << "hipGLGetDevices[" << cudaErr << "]: ";
			switch(cudaErr)
			{
				case hipErrorNoDevice:
					std::cout << "No device found." << std::endl;
					break;
				default:
					std::cout << "Error unknown." << std::endl;
			}
			exit(EXIT_FAILURE);
		}

		cudaErr = cudaGLSetGLDevice(devices[0]);
		if(cudaErr != hipSuccess)
		{
			std::cout << "cudaGLSetGLDevice: ";
			switch(cudaErr)
			{
				case hipErrorInvalidDevice:
					std::cout << "Invalid device." << std::endl;
					break;
				default:
					std::cout << "Device already set." << std::endl;
			}
			exit(EXIT_FAILURE);
		}

		// setup draw buffer
		glGenBuffers(1,&drawBuffer);

		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, drawBuffer);

		glBufferData(GL_PIXEL_UNPACK_BUFFER, dim.x * dim.y * sizeof(Color4), NULL, GL_DYNAMIC_COPY);

		cudaGLRegisterBufferObject(drawBuffer);

		// setup texture
		glEnable(GL_TEXTURE_2D);
		glGenTextures(1,&textureId);
		
		glBindTexture(GL_TEXTURE_2D, textureId);

		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, dim.x, dim.y, 0, GL_RGBA, GL_FLOAT, nullptr);

		glTexParameteri(GL_TEXTURE_2D,GL_TEXTURE_MIN_FILTER,GL_LINEAR);
		glTexParameteri(GL_TEXTURE_2D,GL_TEXTURE_MAG_FILTER,GL_LINEAR);
		
		float fLargest;
		glGetFloatv(GL_MAX_TEXTURE_MAX_ANISOTROPY_EXT, &fLargest);
		glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_MAX_ANISOTROPY_EXT, fLargest);

		// malloc initialize pixel map
		pixelKeyData = thrust::device_vector<PathAggregate>(dim.x*dim.y);
		pixelKeys = thrust::device_vector<uint64_t>(dim.x*dim.y);
		pixelValues = thrust::device_vector<int>(dim.x*dim.y);

		// fill pixel keys and values
		thrust::fill(pixelKeys.begin(), pixelKeys.end(), 0);
		thrust::sequence(pixelValues.begin(), pixelValues.end());
	}

	Renderer::~Renderer()
	{
		glutDestroyWindow(winId);
	}

	void Renderer::moveCamera(const math::vec2 &pos, const math::vec2 &dir)
	{
		DevParams param;
		hipMemcpyFromSymbol(&param, HIP_SYMBOL(devParams), sizeof(DevParams));

		Scene *scene = (Scene*)&param;

		scene->camera.forward = math::rotate(scene->camera.forward, dir.x, scene->camera.up);

		math::vec3 right = math::cross(scene->camera.forward, scene->camera.up);

		scene->camera.forward = math::rotate(scene->camera.forward, dir.y, right);
		scene->camera.up = math::rotate(scene->camera.up, dir.y, right);

		math::vec3 delta = scene->camera.forward * pos.y + right * pos.x;

		scene->camera.position += delta;

		hipMemcpyToSymbol(HIP_SYMBOL(devParams), &param, sizeof(DevParams));

		framesNoMove = 0;
	}

	void Renderer::loadScene(const Scene &scene)
	{
		DevParams params;
		std::memcpy(params.sceneData, &scene, sizeof(Scene));
		params.width = dim.x;
		params.height = dim.y;
		params.samples = dim.z;
		params.paths = thrust::raw_pointer_cast(pixelKeyData.data());
		params.pixelValues = thrust::raw_pointer_cast(pixelValues.data());

		Scene *myScene = (Scene*)&params;
		myScene->camera.aspectRatio = float(dim.x) / float(dim.y);

		hipMemcpyToSymbol(HIP_SYMBOL(devParams), &params, sizeof(DevParams));
	}

	__device__
	math::vec3 get_pixel_dir(const Camera &camera, float ni, float nj)
	{

		math::vec3 dir;
		math::vec3 up;
		float AR;

		math::vec3 cR;
		math::vec3 cU;
		float dist;
		math::vec3 pos;
    
		dir = camera.forward;
		up = camera.up;
		AR = camera.aspectRatio;
		cR = math::cross(dir, up);
		cU = math::cross(cR, dir);
		pos = camera.position;
		dist = math::fastertanfull(camera.horizontalFOV/2.0f);
		
		return math::normalize(dir + dist*(float(nj)*cU + AR*float(ni)*cR));
	}

	__device__
	math::vec3 get_pixel_pos(const Camera &camera, float ni, float nj)
	{

		math::vec3 dir;
		math::vec3 up;
		float AR;

		math::vec3 cR;
		math::vec3 cU;
		float dist;
		math::vec3 pos;

		dir = camera.forward;
		up = camera.up;
		AR = camera.aspectRatio;
		cR = math::cross(dir, up);
		cU = math::cross(cR, dir);
		pos = camera.position;
		dist = math::fastertanfull(camera.horizontalFOV / 2.0f);

		return camera.position + dist*(float(nj)*cU + AR*float(ni)*cR);
	}

	template<int N>
	__device__ inline
	Color4 rayColor(const Ray &r, const Color3 source, Scene *scene, hiprandState state, Path &path);

	template<int N>
	__device__ inline
	Color4 rayColor(const Ray &r, const Color3 source, Scene *scene, hiprandState state, Path &path)
	{
		HitInfo info;
		info.t = FLT_MAX;
		if (scene->intersect(r, info, path))
		{
			Material &mat = scene->materials[info.materialIndex];
			
			// direct illum
			Color3 c = mat.ambient
					+ mat.diffuse * scene->lightPoint(info.point.position, info.point.normal, state, path);

			// indirect illum
			Color3 nSource;
			Ray nr;
			nr.o = info.point.position;

			// Opaque
			if (mat.refractiveIndex == 0)
			{

				Color3 cd = source * mat.diffuse;
				Color3 cs = source * mat.specular;

				float Pd = math::compMax(cd) / math::compMax(source);
				float Ps = math::compMax(cs) / math::compMax(source);

				float P = hiprand_uniform(&state);
				// Diffuse bounce
				if (P < Pd)
				{
					nSource = cd * (1 / Pd);
					nr.d = math::randomHemi(info.point.normal, &state);
				}
				// Specular bounce
				else if (P < Pd + Ps)
				{
					nSource = cs * (1 / Ps);
					nr.d = math::reflect(r.d, info.point.normal);
				}
				// Absorbtion
				else
				{
					//count = MAX_BOUNCES - N; //Maybe set to MAX_BOUNCES - (N-1);
					return Color4(c, 1);
				}
			}
			// Translucent
			else
			{
				float n1, n2;
				math::vec3 norm = info.point.normal;
				if (dot(r.d, norm) < 0)
				{
					n1 = mat.refractiveIndex;
					n2 = 1;
				}
				else
				{
					n1 = 1;
					n2 = mat.refractiveIndex;
					norm *= -1;
				}

				float R = math::rSchlick2(r.d, norm, n1, n2);
				if (hiprand_uniform(&state) >= R)
				{
					nr.d = math::refract(r.d, norm, n1 / n2);
				}
				else
				{
					nr.d = math::reflect(r.d, norm);
				}
			}
			return Color4(c, 1) + rayColor<N - 1>(nr, nSource, scene, state, path);
		}
		//count = MAX_BOUNCES - N;
		return Color4(0, 0, 0, 1);
	}

	template<>
	__device__ inline
	Color4 rayColor<0>(const Ray &r, const Color3 source, Scene *scene, hiprandState state, Path &path)
	{
		//count = MAX_BOUNCES;
		return Color4(0, 0, 0, 1);
	}

	__global__
	void scatterTrace(Color4 *screen, unsigned long seed, uint64_t frames)
	{
		const int width = devParams->width;
		const int height = devParams->height;

		int oldx = blockIdx.x * blockDim.x + threadIdx.x;
		int oldy = blockIdx.y * blockDim.y + threadIdx.y;

		if (oldx >= width || oldy >= height)
		{
			return;
		}

		int oldIndex = oldx + width * oldy;
		Path path;

		int index;
		if (frames == 0)
		{
			devParams->pixelValues[oldIndex] = oldIndex;
			index = oldIndex;
		}
		else
		{
			index = devParams->pixelValues[oldIndex];
		}
		
		int x = index % width;
		int y = index / width;

		hiprandState state;
		hiprand_init(index + seed, 0, 0, &state);

		Scene *scene = (Scene*)devParams;

		float dx = 1.0f / width;
		float dy = 1.0f / height;
		
		float i = 2.0f*(float(x) + hiprand_uniform(&state))*dx - 1.0f;
		float j = 2.0f*(float(y) + hiprand_uniform(&state))*dy - 1.0f;

		Ray r;
		r.o = scene->camera.position;
		r.d = get_pixel_dir(scene->camera, i, -j);

		Color4 contribution = rayColor<MAX_BOUNCES>(r, Color3(1,1,1), scene, state, path);

		// average
		if (frames > 0)
		{
			screen[index] *= (float(frames) / float(frames + 1));
			screen[index] += (contribution / float(frames + 1));
			
			//devParams->paths[oldIndex].add(path);
		}
		// reset
		else
		{
			screen[index] = contribution;
			//devParams->paths[oldIndex].set(path);
		}
	}

	int frameCount = 0;
	int frameMod = 30;
	int pixelMapFrameMod = 60;
	float frameRate = 0.0f;
	int oldElapsedTime = 0;
	int startTime;

	void Renderer::render()
	{
		if (frameCount == 0)
		{
			startTime = glutGet(GLUT_ELAPSED_TIME);
		}
		if (frameCount == 200)
		{
			std::cout << "Time for 200 frames: " << glutGet(GLUT_ELAPSED_TIME) - startTime << std::endl;
		}

		// bind draw buffer to device ptr
		Color4 *screen;
		hipError_t err = cudaGLMapBufferObject((void**)&screen, drawBuffer);
		if (err != hipSuccess)
		{
			std::cerr << "cudaGLMapBufferObject: " << hipGetErrorName(err) << std::endl;
		}

		// call kernel to render pixels then draw to screen
		dim3 block(16, 16);
		dim3 grid((dim.x + block.x - 1) / block.x, (dim.y + block.y - 1) / block.y);

		//std::cout << "frame count: " << framesNoMove << std::endl;
		scatterTrace<<<grid,block>>>(screen,rand(),framesNoMove);
		hipDeviceSynchronize();
		framesNoMove++;

		// unbind draw buffer so openGL can use
		cudaGLUnmapBufferObject(drawBuffer);

		// create texture from draw buffer
		glBindBuffer(GL_PIXEL_UNPACK_BUFFER, drawBuffer);
		glActiveTexture(GL_TEXTURE0);
		glBindTexture(GL_TEXTURE_2D, textureId);

		glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, dim.x, dim.y, GL_RGBA, GL_FLOAT, nullptr);
		GLenum glErr = glGetError();
		if (glErr != GL_NO_ERROR)
		{
			std::cerr << "glTexImage2D: " << gluErrorString(glErr) << std::endl;
		}

		// draw fullscreen quad
		glBegin(GL_QUADS);
			glTexCoord2f( 0, 1.0f);
			glVertex3f(0,0,0);
			glTexCoord2f(0,0);
			glVertex3f(0,1.0f,0);
			glTexCoord2f(1.0f,0);
			glVertex3f(1.0f,1.0f,0);
			glTexCoord2f(1.0f,1.0f);
			glVertex3f(1.0f,0,0);
		glEnd();

		// swap buffers
		glutSwapBuffers();
		glutPostRedisplay();

		// reassign pixels
		if (framesNoMove % pixelMapFrameMod == pixelMapFrameMod - 1)
		{
			/*thrust::copy(pixelKeyData.begin(), pixelKeyData.end(), pixelKeys.begin());
			thrust::stable_sort_by_key(pixelKeys.begin(), pixelKeys.end(), pixelValues.begin());*/
		}

		// update framerate
		if (frameCount % frameMod == 0 && frameCount != 0){
			int elapsed = glutGet(GLUT_ELAPSED_TIME);
			float timeInterval = float(elapsed - oldElapsedTime) / float(1000);
			frameRate = float(frameMod) / timeInterval;
			//printf("Framerate: [%f]\n", frameRate);
			oldElapsedTime = elapsed;
		}
		frameCount++;
	}

}
